
#include <hip/hip_runtime.h>
/*
 * Part of LAKE: Towards a Machine Learning-Assisted Kernel with LAKE
 * Copyright (C) 2022-2024 Henrique Fingler
 * Copyright (C) 2022-2024 Isha Tarte
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

__global__ void hello_kernel(int* inputs, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        inputs[id] = id;
    }
}